#include "hip/hip_runtime.h"
/*******************************************************************************
* Author: Sunil Kumar Yadav
* Date: 1 Aug 2021
* Problem: Parallel reductionnchronization example to calculate summation of array
*          In this examplae using interleaved pair method we can reduce overhead in half
*			in thread block and reduce divergence
********************************************************************************/

#include "hip/hip_runtime.h"
#include ""

#include<string.h>
#include<stdlib.h>
#include"common.h"

// Reducing wrap diveragence using parrallel interleaved pair approch
__global__ void parallel_reduction_interleaved_pair_summation(int* input, int* temp, const int size)
{
	int tid = threadIdx.x;
	int gid = threadIdx.x + blockIdx.x * blockDim.x;

	if (gid > size)
		return;

	for (int offset = blockDim.x / 2; offset > 0; offset /= 2)
	{
		if (tid < offset)
			input[gid] += input[gid + offset];

		__syncthreads();					// wait for all thread in a block to calculate summation
	}
	

	if (tid == 0)							// summation result of each block will be at 0th index
		temp[blockIdx.x] = input[gid];

}


// improving neighbored pair kernel
__global__ void parallel_reduction_neighbored_pair_improved(int* input, int* temp, const int size)
{
	int tid = threadIdx.x;
	int gid = threadIdx.x + blockIdx.x * blockDim.x;

	int* i_data = input + blockIdx.x * blockDim.x;       //local pointer to input with block offset

	if (gid > size)
		return;

	
	for (int offset = 1; offset <= blockDim.x / 2; offset *= 2)
	{
		int index = 2 * offset * tid;

		if (index < blockDim.x)
		{
			i_data[index] += i_data[index + offset];
		}

		__syncthreads();
	}					

	if (tid == 0)							// summation result of each block will be at 0th index
		temp[blockIdx.x] = input[gid];

}


int main()
{
	int size = 1 << 27;     // 128Mb of data
	int block_size = 128;
	int grid_size = (size / block_size);

	dim3 block(block_size);
	dim3 grid(grid_size);

	int mem_size = size * sizeof(int);
	int reduction_array_size = grid.x * sizeof(int);

	int* h_input, * h_temp;
	h_input = (int*)malloc(mem_size);
	h_temp = (int*)malloc(reduction_array_size);

	memset(h_input, 0, mem_size);
	memset(h_temp, 0, reduction_array_size);

	//initialize array with random no.
	initialize_1d_array(h_input, size);

	// calculate result on cpu and validate with GPU
	int cpu_result = 0;
	cpu_result = sum_1d_array(h_input, size);

	int* d_input, * d_temp;
	gpuErrchk(hipMalloc((void**)&d_input, mem_size));
	gpuErrchk(hipMalloc((void**)&d_temp, reduction_array_size));

	gpuErrchk(hipMemset(d_temp, 0, reduction_array_size));

	gpuErrchk(hipMemcpy(d_input, h_input, mem_size, hipMemcpyHostToDevice));


	printf("Kernal launch configuration: block size: %d and grid size: %d\n\n", block.x, grid.x);
	// lets spin the gup
	parallel_reduction_interleaved_pair_summation << <grid, block >> > (d_input, d_temp, size);
	//parallel_reduction_neighbored_pair_improved << <grid, block >> > (d_input, d_temp, size);   // improvement over 26.parallel_reduction
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpy(h_temp, d_temp, reduction_array_size, hipMemcpyDeviceToHost));



	// calculate final result from GPU parallel reduction
	int gpu_result = 0;
	for (int i = 0; i < grid.x; i++)
		gpu_result += h_temp[i];



	printf("Result of CPU calculation: %d and GPU calculation: %d \n", cpu_result, gpu_result);
	if (compare_results(cpu_result, gpu_result))
	{
		printf("Result matches\n");
	}
	else
		printf("Result does not matches\n");


	free(h_input);
	free(h_temp);

	gpuErrchk(hipFree(d_input));
	gpuErrchk(hipFree(d_temp));

	gpuErrchk(hipDeviceReset());
	return 0;
}