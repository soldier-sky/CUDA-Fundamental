#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void print_threadIdx(void)
{
	printf("threadIdx.x: %d, threadIdx.y:%d, threadIdx.z:%d \n", 
		threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void print_thread_details(void)
{
	printf("blockIdx.x: %d, blockIdx.y:%d, blockIdx.z:%d, blockDim.x:%d, blockDim.y:%d, blockDim.z:%d, gridDim.x:%d, gridDim.y:%d, gridDim.z:%d  \n",
		blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}

int main()
{
	int nx, ny;
	nx = 16;
	ny = 16;

	dim3 block(8, 8);
	dim3 grid(nx / block.x, ny / block.y);
	
	//print_threadIdx << <grid, block >> > ();
	print_thread_details <<<grid,block>>>();
	hipDeviceSynchronize();

	hipDeviceReset();

	return 0;
}